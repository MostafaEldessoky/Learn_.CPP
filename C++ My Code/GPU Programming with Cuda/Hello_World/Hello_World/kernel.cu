#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <iostream>
// This is a simple CUDA kernel that prints the block number, thread number, and wrap number
__global__ void print() {
	printf("block number %d -- thread number %d -- wrap number %d\n",blockIdx.x,threadIdx.x,threadIdx.x/32);
}

int main() {

	// Launching the kernel with 1 block and 1 thread
	print << <1, 1 >> > ();
	// Launching the kernel with 1 block and 1024 threads
	print << <1, 1024 >> > ();
	// Launching the kernel with 32 blocks and 1024 threads
	print << <32, 1024 >> > ();


	return 0;
}
