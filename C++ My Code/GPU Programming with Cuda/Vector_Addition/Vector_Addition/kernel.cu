﻿
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>


class wrap_cuda {

	int* a;
	int* b;
	int* c;

	public:

		wrap_cuda(int size)  {
			hipMalloc(&a,size * sizeof(int));
			hipMalloc(&b,size * sizeof(int));
			hipMalloc(&c,size * sizeof(int));
		}

		~wrap_cuda() {
			hipFree(a);
			hipFree(b);
			hipFree(c);
		}

		void copy_to_device(int* host_a, int* host_b, int size) {
			hipMemcpy(a, host_a, size * sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(b, host_b, size * sizeof(int), hipMemcpyHostToDevice);
		}

		void copy_to_host(int* host_c, int size) {
			hipMemcpy(host_c, c, size * sizeof(int), hipMemcpyDeviceToHost);
		}
		template<typename F>
		void exc(int blocks,int threads,F f){
			f <<<blocks,threads>>>(a,b,c,threads);
			hipDeviceSynchronize();
		}

};

// generic pointer void* can take any pointer type
void fn(void* ptr) {
	std::cout << *(int*)ptr << std::endl;
	
}

// Kernel function to add two vectors
__global__ void add_vec(int* a, int* b, int* c,int threads) {
	c[blockIdx.x*threads + threadIdx.x] = a[blockIdx.x * threads + threadIdx.x] + b[blockIdx.x * threads + threadIdx.x];
}


int SIZE = 1024;

int main() {

	int* host_a = new int[SIZE*2];
	int* host_b = new int[SIZE*2];
	int* host_c = new int[SIZE*2];

	wrap_cuda cuda_wrapper(SIZE*2);

	for (int i = 0; i < SIZE*2; ++i) {
		host_a[i] = i;
		host_b[i] = SIZE*2 - i;
	}

	cuda_wrapper.copy_to_device(host_a, host_b, SIZE*2);
	cuda_wrapper.exc(2, SIZE, add_vec);
	cuda_wrapper.copy_to_host(host_c, SIZE*2);

	fn(host_c);

	for (int i = 0; i < SIZE*2; i++) {
		std::cout << *(host_c + i) << " ";
	}
	std::cout << std::endl;

	delete[] host_a;
	delete[] host_b;
	delete[] host_c;

	return 0;
}